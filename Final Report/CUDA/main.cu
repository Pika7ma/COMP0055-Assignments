#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <limits.h>
#include <stdio.h>
#include <sys/time.h>
hipError_t maxPooling(const int n, const int kn_size, float *m, float *m_);

__global__ void maxPoolingKernel1(const int n, const int kn_size, float *dev_m, float *dev_m_tmp) {
    int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset > n * (n - kn_size)) {
        return;
    }
    int x = offset / (n - kn_size + 1);
    int y = offset % (n - kn_size + 1);
    float max_ = -FLT_MAX;
    for (int i = 0; i < kn_size; ++i) {
        if (max_ < dev_m[x * n + y + i]) {
            max_ = dev_m[x * n + y + i];
        }
    }
    dev_m_tmp[offset] = max_;
}

__global__ void maxPoolingKernel2(const int n, const int kn_size, float *dev_m_tmp, float *dev_m_) {
    int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset > (n - kn_size) * (n - kn_size)) {
        return;
    }
    int x = offset / (n - kn_size + 1);
    int y = offset % (n - kn_size + 1);
    float max_ = -FLT_MAX;
    for (int i = 0; i < kn_size; ++i) {
        if (max_ < dev_m_tmp[x * (n - kn_size + 1 + i) + y]) {
            max_ = dev_m_tmp[x * (n - kn_size + 1 + i) + y];
        }
    }
    dev_m_[offset] = max_;
}

int main() {
    const int MAX_N = 4096;
    const int KN_SIZE = 128;

    float *m = new float[MAX_N * MAX_N];
    float *m_ = new float[(MAX_N - KN_SIZE + 1) * (MAX_N - KN_SIZE + 1)];

    // Add vectors in parallel.
    hipError_t cudaStatus = maxPooling(MAX_N, KN_SIZE, m, m_);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "maxPooling failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    getchar();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t maxPooling(const int n, const int kn_size, float *m, float *m_) {
    float *dev_m = 0;
    float *dev_m_tmp = 0;
    float *dev_m_ = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_m, n * n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_m_tmp, n * (n - kn_size + 1) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_m_, (n - kn_size + 1) * (n - kn_size + 1) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_m, m, n * n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    // Launch a kernel on the GPU with one thread for each element.
    const long maxThread = 1024;
    maxPoolingKernel1<<<(long)ceil(n * (n - kn_size + 1) / maxThread), maxThread>>>(n, kn_size, dev_m, dev_m_tmp);


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


    maxPoolingKernel2<<<(long)ceil((n - kn_size + 1) * (n - kn_size + 1) / maxThread), maxThread>>>(n, kn_size, dev_m_tmp, dev_m_);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(m_, dev_m_, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_m);
    hipFree(dev_m_tmp);
    hipFree(dev_m_);

    return cudaStatus;
}
